/* ***************************************************************************************************************
 *
 * VERSAO INICIAL: 13/02/2014
 *
 * ULTIMA ATUALIZACAO: 13/02/2014
 *
 * ***************************************************************************************************************
 *
 * DESCRICAO:
 * 		- Integra o modelo dinamico o oscilador forcado de Duffing;
 *
 *
 * ***************************************************************************************************************
 *
 * 	LIMITACAO:
 * 		- Retorna apenas o estado do sistema no instante final do intervalo de tempo considerado;
 *
 *
 * ***************************************************************************************************************
 *
 * REFERENCIAS:
 *
 * 	http://www.cprogramming.com/tutorial/cfileio.html
 *
 * 	http://stackoverflow.com/questions/11573974/write-to-txt-file
 *
 * ***************************************************************************************************************
 */

#include <math.h>
#include <hip/hip_runtime.h>
	// For the CUDA runtime routines (prefixed with "cuda_")

//#define PI 3.14159265358979323846264338327


/* *************************************************************************************************************** */
/* DESCRICAO:
 * 	- Work out which piece of the global array this thread should operate on;
 * 	- Por ter sido definido como DEVICE, somente pode ser envocado pelo proprio "device", i.e., somente pela GPU;
 * 	- Essa funcao foi definida para facilitar a manipulacao dos indices envolvidos dentro do kernel principal;
 * 	- Utiliza como base o script originalmente desenvolvido para a iteracao do conjunto de Mandelbrot (MATLAB);
 */
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    //	- As entradas serao matrizes;
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}
/* *************************************************************************************************************** */



/* *************************************************************************************************************** */
/* DESCRICAO:
 *  - Main entry point;
 *  - Works out where the current thread should read/write to global memory and calls doIterations to do the actual work.
 */
__global__ void processDuffingTimeEvolution(
                      double * X, 
                      double * Y,
                      double * Z,
                      const double * X0, 
                      const double * Y0,
                      const double * Z0,
                      const double * H,
                      const double * Gamma,
                      const double * Epsilon,
                      const double * Omega,
                      const unsigned int nIntegra,
                      const unsigned int numel ) {
	
	/* **************************************************************************************** */
	// AQUISICAO DOS INDICES REFERENTES AHS THREADS E BLOCOS

	// Work out which thread we are
	size_t const globalThreadIdx = calculateGlobalIndex();

	// If we're off the end, return now
	if (globalThreadIdx >= numel) {
		return;
	}
	/* **************************************************************************************** */
	// INICIALIZACAO DOS PARAMETROS DO SISTEMA DINAMICO

	double epsilon = Epsilon[globalThreadIdx];
	double gamma = Gamma[globalThreadIdx];
	double omega = Omega[globalThreadIdx];

	/* **************************************************************************************** */
	// DEFINE O PASSO DE INCREMENTO DO INTEGRADOR

	const double h = H[globalThreadIdx];

	/* **************************************************************************************** */
	// CARREGA AS CONDICOES INICIAIS PASSADAS AO KERNEL PELO SCRIPT EM MATLAB

	double x0 = X0[globalThreadIdx];
	double y0 = Y0[globalThreadIdx];
	double z0 = Z0[globalThreadIdx];

	/* **************************************************************************************** */
	// INICIALIZA A VARIAVEL DE CONTAGEM PARA A INTEGRACAO DO MODELO DINAMICO

	unsigned int count = 0;

	/* **************************************************************************************** */
	// Inicializacao de condicoes iniciais diversas
	double k1x, k2x, k3x, k4x;
	double k1y, k2y, k3y, k4y;
	double k1z, k2z, k3z, k4z;

	double x, y, z;

	/* **************************************************************************************** */
    // INTEGRA O MODELO DINAMICO

	while ( count < nIntegra ){

       	/* ----------------------------------------------------------------------------------- */
       	// 1o. PASSO:
       	//		- CALCULA OS COEFICIENTES PARCIAIS
       	k1x = h*( y0 );
       	k1y = h*( x0 - pow(x0,3) - epsilon*y0 + gamma*cos( omega*z0 ) );
       	k1z = h*( 1 );

       	k2x = h*( (y0+0.5*k1y) );
       	k2y = h*( (x0+0.5*k1x) - pow((x0+0.5*k1x),3) - epsilon*(y0+0.5*k1y) + gamma*cos( omega*(z0+0.5*k1z) ) );
       	k2z = h*( 1 );

       	k3x = h*( (y0+0.5*k2y) );
       	k3y = h*( (x0+0.5*k2x) - pow((x0+0.5*k2x),3) - epsilon*(y0+0.5*k2y) + gamma*cos( omega*(z0+0.5*k2z) ) );
       	k3z = h*( 1 );

       	k4x = h*( (y0+k3y) );
       	k4y = h*( (x0+k3x) - pow((x0+k3x),3) - epsilon*(y0+k3y) + gamma*cos( omega*(z0+k3z) ) );
       	k4z = h*( 1 );
       	/* ----------------------------------------------------------------------------------- */
       	// 2o. PASSO:
       	//		- ATUALIZA AS VARIAVEIS DE ESTADO
       	x = x0 + (1/6.0)*( k1x + 2*k2x + 2*k3x + k4x );
       	y = y0 + (1/6.0)*( k1y + 2*k2y + 2*k3y + k4y );
       	z = z0 + (1/6.0)*( k1z + 2*k2z + 2*k3z + k4z );
       	/* ----------------------------------------------------------------------------------- */
       	// 3o. PASSO:
       	//		- ATUALIZA AS VARIAVEIS DE ESTADO PARA A PROXIMA ITERACAO
       	x0 = x;
       	y0 = y;
       	z0 = z;
       	/* ----------------------------------------------------------------------------------- */
       	// 4o. PASSO:
       	//		- ATUALIZA O CONTADOR
       	count++;
       	/* ----------------------------------------------------------------------------------- */
	}

   	/* **************************************************************************************** */
	// RETORNA O ESTADO FINAL DO SISTEMA APOS O NUMERO DE ITERACOES MAXIMO CONSIDERADO

	X[globalThreadIdx] = x;
    Y[globalThreadIdx] = y;
    Z[globalThreadIdx] = z;


    /* **************************************************************************************** */


}
/* ************************************************************************************************ */
// EOF: End Of File
/* ************************************************************************************************ */
