/* ***************************************************************************************************************************
 * 
 * VERSAO INICIAL: 23/07/2013
 *
 * ULTIMA ATUALIZACAO: 18/03/2014
 *
 *****************************************************************************************************************************
 * DESCRICAO:
 * 		- Implementa o algoritmo completo de estimacao dos expoentes de Lyapunov pela abordagem ClDyn;
 *
 * 		- Emprega o algoritmo RK4;
 *
 * 		- Analise para o modelo dinamico do oscilador forcado de Duffing;
 *
 * 		- Retorna os vetores que contem os globais e as somas ao final do intervalo de tempo considerado;
 *
 *
 *****************************************************************************************************************************
 * PENDENCIAS E MODIFICACOES:
 * 		- Passar "nMaxItera" e/ou "t_final" como argumento do kernel;
 * 			- Talvez procurar alguma coisa relacionada com aloca��o din�mica
 *
 * 		- Empregar "shared memory":
 * 			- O algoritmo come�a r�pido mas a medida que as itera��es v�o sendo contempladas, o algoritmo
 * 			fica cada vez mais lento atingindo um limiar de, aproximadamente, 0.17 seg por itera��o;
 *
 *
 *****************************************************************************************************************************
 * REFERENCIAS
 * 			[1] https://www.google.com/search?q=error%3A+constant+value+is+not+known+CUDA&ie=UTF-8&sa=Search&channel=fe&client=browser-ubuntu&hl=en
 *
 * 			[2] http://stackoverflow.com/questions/9936376/how-to-defined-a-cuda-shared-memory-with-a-size-known-at-run-time
 *
 * 			[3] http://stackoverflow.com/questions/5531247/allocating-shared-memory/5531640#5531640
 *
 * 			[4] Referencias sobre a constru��o de ondas quadradas em C:
 * 			http://stackoverflow.com/questions/1073606/is-there-a-one-line-function-that-generates-a-triangle-wave
 *
 *
 *****************************************************************************************************************************
 */

#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
	// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime_api.h>



/* ************************************************************************************************************************ */

#define nMaxIteracoes 	800		//( (t_final - t_init) / t_gsr );
#define nIntegra 		50		//( t_gsr / h );

/* ************************************************************************************************************************ */



/* ************************************************************************************************************************ */
/* DESCRICAO:
 * 	- Work out which piece of the global array this thread should operate on;
 * 	- Por ter sido definido como DEVICE, somente pode ser envocado pelo proprio "device", i.e., somente pela GPU;
 * 	- Essa funcao foi definida para facilitar a manipulacao dos indices envolvidos dentro do kernel principal;
 * 	- Utiliza como base o script originalmente desenvolvido para a iteracao do conjunto de Mandelbrot (MATLAB);
 */

__device__ size_t calculateGlobalIndex() {
	// Which block are we?
	size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
	// Which thread are we within the block?
	size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
	// How big is each block?
	//	- As entradas serao matrizes;
	size_t const threadsPerBlock = blockDim.x*blockDim.y;
	// Which thread are we overall?
	return localThreadIdx + globalBlockIndex*threadsPerBlock;
}


/* ************************************************************************************************************************ */
/* ************************************************************************************************************************ */
//															KERNEL															//
/* ************************************************************************************************************************ */
/* ************************************************************************************************************************ */
__global__ void Duffing1989RK4_ClDyn(
		double * outLyap1, 	
		double * outLyap2, 
		double * outLyap3,
		double * vX0,
		double * vY0,
		double * vZ0,
		const double * Gamma,
		const double * Epsilon,
		const double * Omega,
		const unsigned int numElements) {


	/* ******************************************************************************************************************** */
	// AQUISICAO DOS INDICES REFERENTES AHS THREADS E AOS BLOCOS															//
	/* ******************************************************************************************************************** */

	// Work out which thread we are
	size_t const globalThreadIdx = calculateGlobalIndex();

	// If we're off the end, return now
	if (globalThreadIdx >= numElements) {
		return;
	}

	/* ******************************************************************************************************************** */
	// DEFINE A DIMENSAO DO SISTEMA DINAMICO A SER ANALISADO																//
	/* ******************************************************************************************************************** */

	const int dim = 3;

	/* ******************************************************************************************************************** */
	// DEFINE O PASSO DE INCREMENTO DO INTEGRADOR																			//
	/* ******************************************************************************************************************** */

	const double h = 0.01;//H[globalThreadIdx];

	/* ******************************************************************************************************************** */
	// INICIALIZACAO DOS PARAMETROS DO SISTEMA DINAMICO																		//
	/* ******************************************************************************************************************** */

	double epsilon 	= Epsilon[globalThreadIdx];
	double gamma 	= Gamma[globalThreadIdx];
	double omega 	= Omega[globalThreadIdx];

	/* ******************************************************************************************************************** */
	// DEFINICAO DAS CONDICOES INICIAIS E OPCOES DE INTEGRACAO																//
	/* ******************************************************************************************************************** */

	// Instante inicial de integracao
	const double t_init = 0;
	double tempo = 0;

	// Define o instante de tempo final de simulacao
	//const double t_final = 0.5;

	// Define o intervalo de tempo para o qual o sistema (ORIGINAL + CLONES) eh integrado antes da aplicacao do procedimento
	// de reortonormizacao de Gram-Schimidt
	const double t_gsr = 0.5;

	// Define o numero maximo de iteracoes
	//		- Deve ser definido como constante caso seja interessante empregar em indices de vetores;
	//const int nMaxIteracoes = ( (t_final - t_init) / t_gsr );

	// Define quantas iteracoes do algoritmo RK4 serao executadas a cada iteracao
	//const int nIntegra = ( t_gsr / h );

	/* ******************************************************************************************************************** */
	// ALOCA MEMORIA PARA AS VARIAVEIS NECESSARIAS AO CALCULO DO INTEGRADOR 'RK4'											//
	/* ******************************************************************************************************************** */

	// Coeficientes parciais para o sistema ORIGINAL
	double k1y1, k2y1, k3y1, k4y1;
	double k1y2, k2y2, k3y2, k4y2;
	double k1y3, k2y3, k3y3, k4y3;


	// Coeficientes parciais para os sistemas CLONADOS
	double k1y4, k2y4, k3y4, k4y4; 		// clones referente variavel y(1)
	double k1y5, k2y5, k3y5, k4y5;
	double k1y6, k2y6, k3y6, k4y6;

	double k1y7, k2y7, k3y7, k4y7; 		// clones referente variavel y(2)
	double k1y8, k2y8, k3y8, k4y8;
	double k1y9, k2y9, k3y9, k4y9;

	double k1y10, k2y10, k3y10, k4y10; // clones referente variavel y(3)
	double k1y11, k2y11, k3y11, k4y11;
	double k1y12, k2y12, k3y12, k4y12;


	// Coeficientes finais para o sistema ORIGINAL
	double ky1, ky2, ky3;


	// Coeficientes finais para os sistemas CLONADOS
	double ky4, ky5, ky6; 		// clones referente variavel y(1)
	double ky7, ky8, ky9; 		// clones referente variavel y(2)
	double ky10, ky11, ky12;	// clones referente variavel y(3)


	// Variaveis temporarias referente ah atualizacao dos estados
	double Y1, Y2, Y3; 		// sistema ORIGINAL

	double Y4, Y5, Y6; 		// clones referente variavel y(1)
	double Y7, Y8, Y9; 		// clones referente variavel y(2)
	double Y10, Y11, Y12;	// clones referente variavel y(3)

	/* ******************************************************************************************************************** */
	// ALOCA MEMORIA PARA AS VARIAVEIS NECESSARIAS AO PROCEDIMENTO GSR														//
	/* ******************************************************************************************************************** */

	double vk[dim][dim];
	double uk[dim][dim];

	double deltax[dim][dim];

	/* ******************************************************************************************************************** */
	// ALOCA MEMORIA PARA OS VETORES QUE ARMAZENAM OS VALORES DOS EXPOENTES													//
	/* ******************************************************************************************************************** */

	double LyapSoma[dim][nMaxIteracoes + 1];
	double Lyap[dim][nMaxIteracoes + 1];

	LyapSoma[0][0] = 0;
	LyapSoma[1][0] = 0;
	LyapSoma[2][0] = 0;

	Lyap[0][0] = 0;
	Lyap[1][0] = 0;
	Lyap[2][0] = 0;

	/* ******************************************************************************************************************** */
	// ALOCA VARIAVEIS AUXILIARES (contadores, loops, etc)																	//
	/* ******************************************************************************************************************** */

	unsigned int idxIteracoes = 0;
	unsigned int idxIntegra = 0;

	/* ******************************************************************************************************************** */
	// INICIALIZACAO DAS CONDICOES INICIAIS																					//
	/* ******************************************************************************************************************** */

	// Define o valor da perturbacao inicial aplicada aos clones
	const double delta = 0.0001;


	// Sistema ORIGINAL
	double y10 = vX0[globalThreadIdx];
	double y20 = vY0[globalThreadIdx];
	double y30 = vZ0[globalThreadIdx];

	// CLONES REFERENTE PARA AH VARIAVEL y(1)
	double y40 = y10 + delta;
	double y50 = y10;
	double y60 = y10;

	// CLONES REFERENTE PARA AH VARIAVEL y(2)
	double y70 = y20;
	double y80 = y20 + delta;
	double y90 = y20;

	// CLONES REFERENTE PARA AH VARIAVEL y(3)
	double y100 = y30;
	double y110 = y30;
	double y120 = y30 + delta;

	/* ******************************************************************************************************************** */
	// INICIALIZACAO DO MODULO PRINCIPAL																					//
	/* ******************************************************************************************************************** */

	while (idxIteracoes < nMaxIteracoes) {

		/* ================================================================================================================ */
		// INTEGRACAO DO MODELO DINAMICO
		while (idxIntegra < nIntegra) {

			/* ------------------------------------------------------------------------------------------------------------ */
			// 1o. PASSO
			/* ------------------------------------------------------------------------------------------------------------ */

			// 		- COEFICIENTES PARCIAIS PARA O SISTEMA ORIGINAL
			k1y1 = h*( y20 );
			k1y2 = h*( y10 - pow(y10,3) - epsilon*y20 + gamma*cos( omega*y30 ) );
			k1y3 = h*( 1 );

			k2y1 = h*( (y20+0.5*k1y2) );
			k2y2 = h*( (y10+0.5*k1y1) - pow((y10+0.5*k1y1),3) - epsilon*(y20+0.5*k1y2) + gamma*cos( omega*(y30+0.5*k1y3) ) );
			k2y3 = h*( 1 );

			k3y1 = h*( (y20+0.5*k2y2) );
			k3y2 = h*( (y10+0.5*k2y1) - pow((y10+0.5*k2y1),3) - epsilon*(y20+0.5*k2y2) + gamma*cos( omega*(y30+0.5*k2y3) ) );
			k3y3 = h*( 1 );

			k4y1 = h*( (y20+k3y2) );
			k4y2 = h*( (y10+k3y1) - pow((y10+k3y1),3) - epsilon*(y20+k3y2) + gamma*cos( omega*(y30+k3y3) ) );
			k4y3 = h*( 1 );


			// 		- COEFICIENTES PARCIAIS PARA OS CLONES
			k1y4 = h*( y70 );
			k1y5 = h*( y80 );
			k1y6 = h*( y90 );
			k1y7 = h*( y40 - pow(y40,3) - epsilon*y70 + gamma*cos( omega*y100 ) );
			k1y8 = h*( y50 - pow(y50,3) - epsilon*y80 + gamma*cos( omega*y110 ) );
			k1y9 = h*( y60 - pow(y60,3) - epsilon*y90 + gamma*cos( omega*y120 ) );
			k1y10 = h*( 1 );
			k1y11 = h*( 1 );
			k1y12 = h*( 1 );

			k2y4 = h*( (y70+0.5*k1y7) );
			k2y5 = h*( (y80+0.5*k1y8) );
			k2y6 = h*( (y90+0.5*k1y9) );
			k2y7 = h*( (y40+0.5*k1y4) - pow((y40+0.5*k1y4),3) - epsilon*(y70+0.5*k1y7) + gamma*cos( omega*(y100+0.5*k1y10) ) );
			k2y8 = h*( (y50+0.5*k1y5) - pow((y50+0.5*k1y5),3) - epsilon*(y80+0.5*k1y8) + gamma*cos( omega*(y110+0.5*k1y11) ) );
			k2y9 = h*( (y60+0.5*k1y6) - pow((y60+0.5*k1y6),3) - epsilon*(y90+0.5*k1y9) + gamma*cos( omega*(y120+0.5*k1y12) ) );
			k2y10 = h*( 1 );
			k2y11 = h*( 1 );
			k2y12 = h*( 1 );

			k3y4 = h*( (y70+0.5*k2y7) );
			k3y5 = h*( (y80+0.5*k2y8) );
			k3y6 = h*( (y90+0.5*k2y9) );
			k3y7 = h*( (y40+0.5*k2y4) - pow((y40+0.5*k2y4),3) - epsilon*(y70+0.5*k2y7) + gamma*cos( omega*(y100+0.5*k2y10) ) );
			k3y8 = h*( (y50+0.5*k2y5) - pow((y50+0.5*k2y5),3) - epsilon*(y80+0.5*k2y8) + gamma*cos( omega*(y110+0.5*k2y11) ) );
			k3y9 = h*( (y60+0.5*k2y6) - pow((y60+0.5*k2y6),3) - epsilon*(y90+0.5*k2y9) + gamma*cos( omega*(y120+0.5*k2y12) ) );
			k3y10 = h*( 1 );
			k3y11 = h*( 1 );
			k3y12 = h*( 1 );

			k4y4 = h*( (y70+k3y7) );
			k4y5 = h*( (y80+k3y8) );
			k4y6 = h*( (y90+k3y9) );
			k4y7 = h*( (y40+k3y4) - pow((y40+k3y4),3) - epsilon*(y70+k3y7) + gamma*cos( omega*(y100+k3y10) ) );
			k4y8 = h*( (y50+k3y5) - pow((y50+k3y5),3) - epsilon*(y80+k3y8) + gamma*cos( omega*(y110+k3y11) ) );
			k4y9 = h*( (y60+k3y6) - pow((y60+k3y6),3) - epsilon*(y90+k3y9) + gamma*cos( omega*(y120+k3y12) ) );
			k4y10 = h*( 1 );
			k4y11 = h*( 1 );
			k4y12 = h*( 1 );

			/* ------------------------------------------------------------------------------------------------------------ */
			// 2o. PASSO
			/* ------------------------------------------------------------------------------------------------------------ */

			// 		- COEFICIENTES PARA O SISTEMA ORIGINAL
			ky1 = (1/6.0)*( k1y1 + 2*k2y1 + 2*k3y1 + k4y1 );
			ky2 = (1/6.0)*( k1y2 + 2*k2y2 + 2*k3y2 + k4y2 );
			ky3 = (1/6.0)*( k1y3 + 2*k2y3 + 2*k3y3 + k4y3 );

			// 		- COEFICIENTES PARA OS CLONES
			ky4 = (1/6.0)*( k1y4 + 2*k2y4 + 2*k3y4 + k4y4 );
			ky5 = (1/6.0)*( k1y5 + 2*k2y5 + 2*k3y5 + k4y5 );
			ky6 = (1/6.0)*( k1y6 + 2*k2y6 + 2*k3y6 + k4y6 );

			ky7 = (1/6.0)*( k1y7 + 2*k2y7 + 2*k3y7 + k4y7 );
			ky8 = (1/6.0)*( k1y8 + 2*k2y8 + 2*k3y8 + k4y8 );
			ky9 = (1/6.0)*( k1y9 + 2*k2y9 + 2*k3y9 + k4y9 );

			ky10 = (1/6.0)*( k1y10 + 2*k2y10 + 2*k3y10 + k4y10 );
			ky11 = (1/6.0)*( k1y11 + 2*k2y11 + 2*k3y11 + k4y11 );
			ky12 = (1/6.0)*( k1y12 + 2*k2y12 + 2*k3y12 + k4y12 );

			/* ------------------------------------------------------------------------------------------------------------ */
			// 3o. PASSO:
			/* ------------------------------------------------------------------------------------------------------------ */

			//   - Atualizacao do sistema (aplicacao do PASSO de Euler);
			// SISTEMA ORIGINAL
			Y1 = y10 + ky1;
			Y2 = y20 + ky2;
			Y3 = y30 + ky3;

			// CLONES
			Y4 = y40 + ky4;
			Y5 = y50 + ky5;
			Y6 = y60 + ky6;

			Y7 = y70 + ky7;
			Y8 = y80 + ky8;
			Y9 = y90 + ky9;

			Y10 = y100 + ky10;
			Y11 = y110 + ky11;
			Y12 = y120 + ky12;

			/* ------------------------------------------------------------------------------------------------------------ */
			// ATUALIZACAO DAS CONDICOES PARA A PROXIMA ITERACAO
			/* ------------------------------------------------------------------------------------------------------------ */

			y10 = Y1;
			y20 = Y2;
			y30 = Y3;

			y40 = Y4;
			y50 = Y5;
			y60 = Y6;

			y70 = Y7;
			y80 = Y8;
			y90 = Y9;

			y100 = Y10;
			y110 = Y11;
			y120 = Y12;

			/* ------------------------------------------------------------------------------------------------------------ */
			// Atualiza o contador
			/* ------------------------------------------------------------------------------------------------------------ */

			idxIntegra++;

			/* ------------------------------------------------------------------------------------------------------------ */

		} // FINALIZACAO DA INTEGRACAO DO MODELO DINAMICO


		/* ================================================================================================================ */
		// Salva o instante de tempo final
		/* ================================================================================================================ */

		tempo = tempo + t_gsr;

		/* ================================================================================================================ */
		// CALCULA O VETOR DIFERENCA DE ESTADOS
		/* ================================================================================================================ */

		// Diferenca de estado referente ah variavel y(1)
		deltax[0][0] = Y1 - Y4;
		deltax[0][1] = Y1 - Y5;
		deltax[0][2] = Y1 - Y6;

		// Diferenca de estado referente ah variavel y(2)
		deltax[1][0] = Y2 - Y7;
		deltax[1][1] = Y2 - Y8;
		deltax[1][2] = Y2 - Y9;

		// Diferenca de estado referente ah variavel y(2)
		deltax[2][0] = Y3 - Y10;
		deltax[2][1] = Y3 - Y11;
		deltax[2][2] = Y3 - Y12;

		/* ================================================================================================================ */
		// PROCEDIMENTO DE REORTONORMALIZACAO DE GRAM-SCHIMIDT
		/* ================================================================================================================ */

		// Expoente Lyap1
		vk[0][0] = deltax[0][0];
		vk[1][0] = deltax[1][0];
		vk[2][0] = deltax[2][0];

		double Norm1 = sqrt( pow(vk[0][0],2)+pow(vk[1][0],2)+pow(vk[2][0],2) );

		uk[0][0] = vk[0][0] / Norm1;
		uk[1][0] = vk[1][0] / Norm1;
		uk[2][0] = vk[2][0] / Norm1;

		// Expoente Lyap2
		double pdi1 = ( uk[0][0]*deltax[0][1]+uk[1][0]*deltax[1][1]+uk[2][0]*deltax[2][1]) / ( uk[0][0]*uk[0][0]+uk[1][0]*uk[1][0]+uk[2][0]*uk[2][0] );

		vk[0][1] = deltax[0][1] - pdi1 * uk[0][0];
		vk[1][1] = deltax[1][1] - pdi1 * uk[1][0];
		vk[2][1] = deltax[2][1] - pdi1 * uk[2][0];

		double Norm2 = sqrt( pow(vk[0][1],2)+pow(vk[1][1],2)+pow(vk[2][1],2) );

		uk[0][1] = vk[0][1] / Norm2;
		uk[1][1] = vk[1][1] / Norm2;
		uk[2][1] = vk[2][1] / Norm2;

		// Expoente Lyap3
		double pdi2 = ( uk[0][0]*deltax[0][2]+uk[1][0]*deltax[1][2]+uk[2][0]*deltax[2][2]) / ( uk[0][0]*uk[0][0]+uk[1][0]*uk[1][0]+uk[2][0]*uk[2][0] );
		double pdi3 = ( uk[0][1]*deltax[0][2]+uk[1][1]*deltax[1][2]+uk[2][1]*deltax[2][2]) / ( uk[0][1]*uk[0][1]+uk[1][1]*uk[1][1]+uk[2][1]*uk[2][1] );

		vk[0][2] = deltax[0][2] - pdi2 * uk[0][0] - pdi3 * uk[0][1];
		vk[1][2] = deltax[1][2] - pdi2 * uk[1][0] - pdi3 * uk[1][1];
		vk[2][2] = deltax[2][2] - pdi2 * uk[2][0] - pdi3 * uk[2][1];

		double Norm3 = sqrt( pow(vk[0][2],2)+pow(vk[1][2],2)+pow(vk[2][2],2) );

		uk[0][2] = vk[0][2] / Norm3;
		uk[1][2] = vk[1][2] / Norm3;
		uk[2][2] = vk[2][2] / Norm3;

		/* ================================================================================================================ */
		// CALCULO DOS EXPOENTES DE LYAPUNOV
		/* ================================================================================================================ */

		// Somatorio das normas para ser utilizado no calculo do expoentes globais
		LyapSoma[0][idxIteracoes + 1] = LyapSoma[0][idxIteracoes] + log(Norm1 / delta);
		LyapSoma[1][idxIteracoes + 1] = LyapSoma[1][idxIteracoes] + log(Norm2 / delta);
		LyapSoma[2][idxIteracoes + 1] = LyapSoma[2][idxIteracoes] + log(Norm3 / delta);

		// Calcula os expoentes globais
		Lyap[0][idxIteracoes + 1] = (1 / (tempo - t_init)) * LyapSoma[0][idxIteracoes + 1];
		Lyap[1][idxIteracoes + 1] = (1 / (tempo - t_init)) * LyapSoma[1][idxIteracoes + 1];
		Lyap[2][idxIteracoes + 1] = (1 / (tempo - t_init)) * LyapSoma[2][idxIteracoes + 1];

		/* ================================================================================================================ */
		// INICIALIZACAO DAS CONDICOES INICIAIS PARA A PROXIMA ITERACAO
		/* ================================================================================================================ */

		// Sistema ORIGINAL
		y10 = Y1;
		y20 = Y2;
		y30 = Y3;

		// CLONES REFERENTE PARA AH VARIAVEL y(1)
		y40 = Y1 + delta*uk[0][0];
		y50 = Y1 + delta*uk[0][1];
		y60 = Y1 + delta*uk[0][2];

		// CLONES REFERENTE PARA AH VARIAVEL y(2)
		y70 = Y2 + delta*uk[1][0];
		y80 = Y2 + delta*uk[1][1];
		y90 = Y2 + delta*uk[1][2];

		// CLONES REFERENTE PARA AH VARIAVEL y(3)
		y100 = Y3 + delta*uk[2][0];
		y110 = Y3 + delta*uk[2][1];
		y120 = Y3 + delta*uk[2][2];

		/* ================================================================================================================ */
		// Atualizacoes para a iteracao seguinte
		/* ================================================================================================================ */

		idxIteracoes++;
		idxIntegra = 0;

		/* ================================================================================================================ */

	} // FIM DO LOOP REFERENTES AHS ITERACOES

	/* ******************************************************************************************************************** */

	__syncthreads();

	/* ******************************************************************************************************************** */
	// RETORNO DO KERNEL: EXPOENTE GLOBAL																					//
	/* ******************************************************************************************************************** */

	for (unsigned int ii = 0; ii <= nMaxIteracoes; ii++){

		outLyap1[globalThreadIdx*nMaxIteracoes + ii] = Lyap[0][ii];
		outLyap2[globalThreadIdx*nMaxIteracoes + ii] = Lyap[1][ii];
		outLyap3[globalThreadIdx*nMaxIteracoes + ii] = Lyap[2][ii];

	}

	/* ******************************************************************************************************************** */

} // FIM DO KERNEL
/* ************************************************************************************************************************ */
