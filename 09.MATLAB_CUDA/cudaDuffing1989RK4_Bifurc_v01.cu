/*
 * Versao inicial: 10/10/2013
 *
 * Ultima atualizacao: 20/04/2015
 *
 ***********************************************************************************************************************
 * DESCRICAO:
 * 		- Implementa o algoritmo completo de estimacao dos expoentes de Lyapunov pela abordagem ClDyn;
 *
 * 		- Emprega o algoritmo de integracao numerica RK4;
 *
 * 		- Analise para o modelo dinamico do oscilador forcado de Duffing;
 *
 * 		- Retorna os expoentes globais ao final de 't_final';
 *
 *
 ************************************************************************************************************************
 * OBSERVACOES:
 * 		- Esse codigo eh o mesmo daquele usado para a estimacao das Estruturas Lagrangianas Coerentes,
 * 		A MENOS DE MODIFICACOES NO RETORNO DO KERNEL;
 * 		
 * 		- Ao inves de armazenar a evolucao dos expoentes, somente eh acumulada a soma dos expoentes. Isso
 * 		eh necessario para economizar memoria; 
 *
 *
 ************************************************************************************************************************
 * REFERENCIAS
 * 			[1] https://www.google.com/search?q=error%3A+constant+value+is+not+known+CUDA&ie=UTF-8&sa=Search&channel=fe&client=browser-ubuntu&hl=en
 *
 * 			[2] http://stackoverflow.com/questions/9936376/how-to-defined-a-cuda-shared-memory-with-a-size-known-at-run-time
 *
 * 			[3] http://stackoverflow.com/questions/5531247/allocating-shared-memory/5531640#5531640
 *
 * 			[4] Referencias sobre a construcao de ondas quadradas em C:
 * 			http://stackoverflow.com/questions/1073606/is-there-a-one-line-function-that-generates-a-triangle-wave
 *
 *
 ************************************************************************************************************************
 */

#include <math.h>
#include <hip/hip_runtime.h>
	// For the CUDA runtime routines (prefixed with "cuda_")

//#define PI 3.14159265358979323846264338327




/* DESCRICAO:
 * 	- Work out which piece of the global array this thread should operate on;
 * 	- Por ter sido definido como DEVICE, somente pode ser envocado pelo proprio "device", i.e., somente pela GPU;
 * 	- Essa funcao foi definida para facilitar a manipulacao dos indices envolvidos dentro do kernel principal;
 * 	- Utiliza como base o script originalmente desenvolvido para a iteracao do conjunto de Mandelbrot (MATLAB);
 */
__device__ size_t calculateGlobalIndex() {
	// Which block are we?
	size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
	// Which thread are we within the block?
	size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
	// How big is each block?
	size_t const threadsPerBlock = blockDim.x * blockDim.y;
	// Which thread are we overall?
	return localThreadIdx + globalBlockIndex * threadsPerBlock;
}




/* DESCRICAO:
 *  - Main entry point;
 *  - Works out where the current thread should read/write to global memory and calls doIterations to do the actual work.
 */
__global__ void processDuffing1989RK4_Bifurc(
		double * outLyap1, 	double * outLyap2, double * outLyap3,
		const double * Y0_var1, const double * Y0_var2, const double * Y0_var3,
		const double * Y0_var4, const double * Y0_var5, const double * Y0_var6,
		const double * Y0_var7, const double * Y0_var8, const double * Y0_var9,
		const double * Y0_var10, const double * Y0_var11, const double * Y0_var12,
		const double * H,
		const double * Gamma,
		const double * Epsilon,
		const double * Omega,
		const unsigned int numel) {


	/* **************************************************************************************** */
	// AQUISICAO DOS INDICES REFERENTES AHS THREADS E BLOCOS
	// Work out which thread we are
	size_t const globalThreadIdx = calculateGlobalIndex();

	// If we're off the end, return now
	if (globalThreadIdx >= numel) {
		return;
	}
	/* **************************************************************************************** */


	/* **************************************************************************************** */
	// INICIALIZACAO DOS PARAMETROS DO SISTEMA DINAMICO
	double epsilon = Epsilon[globalThreadIdx];
	double gamma = Gamma[globalThreadIdx];
	double omega = Omega[globalThreadIdx];
	/* **************************************************************************************** */
	// DEFINE A DIMENSAO DO SISTEMA DINAMICO A SER ANALISADO
	//	- Em se tratando da metodologia TanMap, o numero total de equacoes a ser integrado eh,
	//	obrigatoriamente, igual ah "dim*(dim+1)";
	const int dim = 3;
	/* **************************************************************************************** */
	// DEFINICAO DAS CONDICOES INICIAIS E OPCOES DE INTEGRACAO
	// Define o valor da perturbacao inicial aplicada aos clones
	const double delta = 0.0001;

	// Instante inicial de integracao
	const double t_init = 0;
	double tempo = 0;

	// Define o instante de tempo final de simulacao
	const double t_final = 10000.0;

	// Define o intervalo de tempo para o qual o sistema (ORIGINAL + CLONES) eh integrado
	const double t_gsr = 0.5;

	// Define o passo de incremento do tempo
	//		- Em outros scripts, esse parametro eh definido como 'passo';
	const double h = H[globalThreadIdx];

	// Define o numero maximo de iteracoes
	//		- Deve ser definido como constante caso seja interessante empregar em indices de vetores;
	const int nMaxIteracoes = ((t_final - t_init) / t_gsr);

	// Define quantas iteracoes do algoritmo RK4 serao executadas a cada iteracao
	const int nIntegra = t_gsr / h;
	/* **************************************************************************************** */
	// ALOCA MEMORIA PARA AS VARIAVEIS NECESSARIAS AO CALCULO DO INTEGRADOR 'RK4'
	// Coeficientes parciais para o sistema ORIGINAL
	double k1y1, k2y1, k3y1, k4y1;
	double k1y2, k2y2, k3y2, k4y2;
	double k1y3, k2y3, k3y3, k4y3;

	// Coeficientes parciais para os sistemas CLONADOS
	double k1y4, k2y4, k3y4, k4y4; 		// clones referente variavel y(1)
	double k1y5, k2y5, k3y5, k4y5;
	double k1y6, k2y6, k3y6, k4y6;

	double k1y7, k2y7, k3y7, k4y7; 		// clones referente variavel y(2)
	double k1y8, k2y8, k3y8, k4y8;
	double k1y9, k2y9, k3y9, k4y9;

	double k1y10, k2y10, k3y10, k4y10; // clones referente variavel y(3)
	double k1y11, k2y11, k3y11, k4y11;
	double k1y12, k2y12, k3y12, k4y12;

	// Coeficientes finais para o sistema ORIGINAL
	double ky1, ky2, ky3;

	// Coeficientes finais para os sistemas CLONADOS
	double ky4, ky5, ky6; 			// clones referente variavel y(1)
	double ky7, ky8, ky9; 			// clones referente variavel y(2)
	double ky10, ky11, ky12;		// clones referente variavel y(3)

	// Variaveis temporarias referente ah atualizacao dos estados
	double Y1, Y2, Y3; 				// sistema ORIGINAL

	double Y4, Y5, Y6; 				// clones referente variavel y(1)
	double Y7, Y8, Y9; 				// clones referente variavel y(2)
	double Y10, Y11, Y12;			// clones referente variavel y(3)
	/* **************************************************************************************** */
	// ALOCA MEMORIA PARA AS VARIAVEIS NECESSARIAS AO PROCEDIMENTO GSR
	double vk[dim][dim];
	double uk[dim][dim];

	double deltax[dim][dim];
	/* **************************************************************************************** */
	// ALOCA MEMORIA PARA OS VETORES QUE ARMAZENAM OS VALORES DOS EXPOENTES
	//double LyapLocal[dim][nMaxIteracoes + 1];
	//double LyapSoma[dim][nMaxIteracoes + 1];
	//double Lyap[dim][nMaxIteracoes + 1];
	double LyapSoma[dim][1];
	//double Lyap[dim][0];

	//LyapLocal[0][0] = 0;
	//LyapLocal[1][0] = 0;
	//LyapLocal[2][0] = 0;

	LyapSoma[0][0] = 0;
	LyapSoma[1][0] = 0;
	LyapSoma[2][0] = 0;

	//Lyap[0][0] = 0;
	//Lyap[1][0] = 0;
	//Lyap[2][0] = 0;
	/* **************************************************************************************** */
	// ALOCA VARIAVEIS AUXILIARES (contadores, loops, etc)
	unsigned int idxIteracoes = 0;
	unsigned int idxIntegra = 0;
	/* **************************************************************************************** */


	/* **************************************************************************************** */
	// INICIALIZACAO DAS CONDICOES INICIAIS

	// Sistema ORIGINAL
	double y10 = Y0_var1[globalThreadIdx];
	double y20 = Y0_var2[globalThreadIdx];
	double y30 = Y0_var3[globalThreadIdx];

	// CLONES REFERENTE PARA AH VARIAVEL y(1)
	double y40 = Y0_var4[globalThreadIdx];
	double y50 = Y0_var5[globalThreadIdx];
	double y60 = Y0_var6[globalThreadIdx];

	// CLONES REFERENTE PARA AH VARIAVEL y(2)
	double y70 = Y0_var7[globalThreadIdx];
	double y80 = Y0_var8[globalThreadIdx];
	double y90 = Y0_var9[globalThreadIdx];

	// CLONES REFERENTE PARA AH VARIAVEL y(3)
	double y100 = Y0_var10[globalThreadIdx];
	double y110 = Y0_var11[globalThreadIdx];
	double y120 = Y0_var12[globalThreadIdx];
	/* **************************************************************************************** */


	/* **************************************************************************************** */
	while (idxIteracoes < nMaxIteracoes) {
		/* ************************************************************************************ */
		// INTEGRACAO DO MODELO DINAMICO
		while (idxIntegra < nIntegra) {
			/* -------------------------------------------------------------------------------- */
			// 1o. PASSO
			// 		- COEFICIENTES PARCIAIS PARA O SISTEMA ORIGINAL
			k1y1 = h*( y20 );
			k1y2 = h*( y10 - pow(y10,3) - epsilon*y20 + gamma*cos( omega*y30 ) );
			k1y3 = h*( 1 );

			k2y1 = h*( (y20+0.5*k1y2) );
			k2y2 = h*( (y10+0.5*k1y1) - pow((y10+0.5*k1y1),3) - epsilon*(y20+0.5*k1y2) + gamma*cos( omega*(y30+0.5*k1y3) ) );
			k2y3 = h*( 1 );

			k3y1 = h*( (y20+0.5*k2y2) );
			k3y2 = h*( (y10+0.5*k2y1) - pow((y10+0.5*k2y1),3) - epsilon*(y20+0.5*k2y2) + gamma*cos( omega*(y30+0.5*k2y3) ) );
			k3y3 = h*( 1 );

			k4y1 = h*( (y20+k3y2) );
			k4y2 = h*( (y10+k3y1) - pow((y10+k3y1),3) - epsilon*(y20+k3y2) + gamma*cos( omega*(y30+k3y3) ) );
			k4y3 = h*( 1 );


			// 		- COEFICIENTES PARCIAIS PARA OS CLONES
			k1y4 = h*( y70 );
			k1y5 = h*( y80 );
			k1y6 = h*( y90 );
			k1y7 = h*( y40 - pow(y40,3) - epsilon*y70 + gamma*cos( omega*y100 ) );
			k1y8 = h*( y50 - pow(y50,3) - epsilon*y80 + gamma*cos( omega*y110 ) );
			k1y9 = h*( y60 - pow(y60,3) - epsilon*y90 + gamma*cos( omega*y120 ) );
			k1y10 = h*( 1 );
			k1y11 = h*( 1 );
			k1y12 = h*( 1 );

			k2y4 = h*( (y70+0.5*k1y7) );
			k2y5 = h*( (y80+0.5*k1y8) );
			k2y6 = h*( (y90+0.5*k1y9) );
			k2y7 = h*( (y40+0.5*k1y4) - pow((y40+0.5*k1y4),3) - epsilon*(y70+0.5*k1y7) + gamma*cos( omega*(y100+0.5*k1y10) ) );
			k2y8 = h*( (y50+0.5*k1y5) - pow((y50+0.5*k1y5),3) - epsilon*(y80+0.5*k1y8) + gamma*cos( omega*(y110+0.5*k1y11) ) );
			k2y9 = h*( (y60+0.5*k1y6) - pow((y60+0.5*k1y6),3) - epsilon*(y90+0.5*k1y9) + gamma*cos( omega*(y120+0.5*k1y12) ) );
			k2y10 = h*( 1 );
			k2y11 = h*( 1 );
			k2y12 = h*( 1 );

			k3y4 = h*( (y70+0.5*k2y7) );
			k3y5 = h*( (y80+0.5*k2y8) );
			k3y6 = h*( (y90+0.5*k2y9) );
			k3y7 = h*( (y40+0.5*k2y4) - pow((y40+0.5*k2y4),3) - epsilon*(y70+0.5*k2y7) + gamma*cos( omega*(y100+0.5*k2y10) ) );
			k3y8 = h*( (y50+0.5*k2y5) - pow((y50+0.5*k2y5),3) - epsilon*(y80+0.5*k2y8) + gamma*cos( omega*(y110+0.5*k2y11) ) );
			k3y9 = h*( (y60+0.5*k2y6) - pow((y60+0.5*k2y6),3) - epsilon*(y90+0.5*k2y9) + gamma*cos( omega*(y120+0.5*k2y12) ) );
			k3y10 = h*( 1 );
			k3y11 = h*( 1 );
			k3y12 = h*( 1 );

			k4y4 = h*( (y70+k3y7) );
			k4y5 = h*( (y80+k3y8) );
			k4y6 = h*( (y90+k3y9) );
			k4y7 = h*( (y40+k3y4) - pow((y40+k3y4),3) - epsilon*(y70+k3y7) + gamma*cos( omega*(y100+k3y10) ) );
			k4y8 = h*( (y50+k3y5) - pow((y50+k3y5),3) - epsilon*(y80+k3y8) + gamma*cos( omega*(y110+k3y11) ) );
			k4y9 = h*( (y60+k3y6) - pow((y60+k3y6),3) - epsilon*(y90+k3y9) + gamma*cos( omega*(y120+k3y12) ) );
			k4y10 = h*( 1 );
			k4y11 = h*( 1 );
			k4y12 = h*( 1 );
			/* -------------------------------------------------------------------------------- */
			// 2o. PASSO
			// 		- COEFICIENTES PARA O SISTEMA ORIGINAL
			ky1 = (1/6.0)*( k1y1 + 2*k2y1 + 2*k3y1 + k4y1 );
			ky2 = (1/6.0)*( k1y2 + 2*k2y2 + 2*k3y2 + k4y2 );
			ky3 = (1/6.0)*( k1y3 + 2*k2y3 + 2*k3y3 + k4y3 );

			// 		- COEFICIENTES PARA OS CLONES
			ky4 = (1/6.0)*( k1y4 + 2*k2y4 + 2*k3y4 + k4y4 );
			ky5 = (1/6.0)*( k1y5 + 2*k2y5 + 2*k3y5 + k4y5 );
			ky6 = (1/6.0)*( k1y6 + 2*k2y6 + 2*k3y6 + k4y6 );

			ky7 = (1/6.0)*( k1y7 + 2*k2y7 + 2*k3y7 + k4y7 );
			ky8 = (1/6.0)*( k1y8 + 2*k2y8 + 2*k3y8 + k4y8 );
			ky9 = (1/6.0)*( k1y9 + 2*k2y9 + 2*k3y9 + k4y9 );

			ky10 = (1/6.0)*( k1y10 + 2*k2y10 + 2*k3y10 + k4y10 );
			ky11 = (1/6.0)*( k1y11 + 2*k2y11 + 2*k3y11 + k4y11 );
			ky12 = (1/6.0)*( k1y12 + 2*k2y12 + 2*k3y12 + k4y12 );
			/* -------------------------------------------------------------------------------- */
			// 3o. PASSO:
			//   - Atualizacao do sistema (aplicacao do PASSO de Euler);
			// SISTEMA ORIGINAL
			Y1 = y10 + ky1;
			Y2 = y20 + ky2;
			Y3 = y30 + ky3;

			// CLONES
			Y4 = y40 + ky4;
			Y5 = y50 + ky5;
			Y6 = y60 + ky6;

			Y7 = y70 + ky7;
			Y8 = y80 + ky8;
			Y9 = y90 + ky9;

			Y10 = y100 + ky10;
			Y11 = y110 + ky11;
			Y12 = y120 + ky12;
			/* -------------------------------------------------------------------------------- */
			// ATUALIZACAO DAS CONDICOES PARA A PROXIMA ITERACAO
			y10 = Y1;
			y20 = Y2;
			y30 = Y3;

			y40 = Y4;
			y50 = Y5;
			y60 = Y6;

			y70 = Y7;
			y80 = Y8;
			y90 = Y9;

			y100 = Y10;
			y110 = Y11;
			y120 = Y12;
			/* -------------------------------------------------------------------------------- */
			// Atualiza o contador
			idxIntegra++;
			/* -------------------------------------------------------------------------------- */
		} // FINALIZACAO DA INTEGRACAO DO MODELO DINAMICO
		/* **************************************************************************************************** */


		/* **************************************************************************************************** */
		// Salva o instante de tempo final
		tempo = tempo + t_gsr;
		/* **************************************************************************************************** */
		// CALCULA O VETOR DIFERENCA DE ESTADOS

		// Diferenca de estado referente ah variavel y(1)
		deltax[0][0] = Y1 - Y4;
		deltax[0][1] = Y1 - Y5;
		deltax[0][2] = Y1 - Y6;

		// Diferenca de estado referente ah variavel y(2)
		deltax[1][0] = Y2 - Y7;
		deltax[1][1] = Y2 - Y8;
		deltax[1][2] = Y2 - Y9;

		// Diferenca de estado referente ah variavel y(2)
		deltax[2][0] = Y3 - Y10;
		deltax[2][1] = Y3 - Y11;
		deltax[2][2] = Y3 - Y12;
		/* **************************************************************************************************** */
		// PROCEDIMENTO DE REORTONORMALIZACAO DE GRAM-SCHIMIDT

		// Expoente Lyap1
		vk[0][0] = deltax[0][0];
		vk[1][0] = deltax[1][0];
		vk[2][0] = deltax[2][0];

		double Norm1 = sqrt( pow(vk[0][0],2)+pow(vk[1][0],2)+pow(vk[2][0],2) );

		uk[0][0] = vk[0][0] / Norm1;
		uk[1][0] = vk[1][0] / Norm1;
		uk[2][0] = vk[2][0] / Norm1;

		// Expoente Lyap2
		double pdi1 = ( uk[0][0]*deltax[0][1]+uk[1][0]*deltax[1][1]+uk[2][0]*deltax[2][1]) / ( uk[0][0]*uk[0][0]+uk[1][0]*uk[1][0]+uk[2][0]*uk[2][0] );

		vk[0][1] = deltax[0][1] - pdi1 * uk[0][0];
		vk[1][1] = deltax[1][1] - pdi1 * uk[1][0];
		vk[2][1] = deltax[2][1] - pdi1 * uk[2][0];

		double Norm2 = sqrt( pow(vk[0][1],2)+pow(vk[1][1],2)+pow(vk[2][1],2) );

		uk[0][1] = vk[0][1] / Norm2;
		uk[1][1] = vk[1][1] / Norm2;
		uk[2][1] = vk[2][1] / Norm2;

		// Expoente Lyap3
		double pdi2 = ( uk[0][0]*deltax[0][2]+uk[1][0]*deltax[1][2]+uk[2][0]*deltax[2][2]) / ( uk[0][0]*uk[0][0]+uk[1][0]*uk[1][0]+uk[2][0]*uk[2][0] );
		double pdi3 = ( uk[0][1]*deltax[0][2]+uk[1][1]*deltax[1][2]+uk[2][1]*deltax[2][2]) / ( uk[0][1]*uk[0][1]+uk[1][1]*uk[1][1]+uk[2][1]*uk[2][1] );

		vk[0][2] = deltax[0][2] - pdi2 * uk[0][0] - pdi3 * uk[0][1];
		vk[1][2] = deltax[1][2] - pdi2 * uk[1][0] - pdi3 * uk[1][1];
		vk[2][2] = deltax[2][2] - pdi2 * uk[2][0] - pdi3 * uk[2][1];

		double Norm3 = sqrt( pow(vk[0][2],2)+pow(vk[1][2],2)+pow(vk[2][2],2) );

		uk[0][2] = vk[0][2] / Norm3;
		uk[1][2] = vk[1][2] / Norm3;
		uk[2][2] = vk[2][2] / Norm3;
		/* ************************************************************************************ */
		// CALCULO DOS EXPOENTES DE LYAPUNOV
		// Calcula os expoentes locais
		//LyapLocal[0][idxIteracoes + 1] = (1 / t_gsr) * log(Norm1 / delta);
		//LyapLocal[1][idxIteracoes + 1] = (1 / t_gsr) * log(Norm2 / delta);
		//LyapLocal[2][idxIteracoes + 1] = (1 / t_gsr) * log(Norm3 / delta);

		// Somatorio das normas para ser utilizado no calculo do expoentes globais
		LyapSoma[0][0] = LyapSoma[0][0] + log(Norm1 / delta);
		LyapSoma[1][0] = LyapSoma[1][0] + log(Norm2 / delta);
		LyapSoma[2][0] = LyapSoma[2][0] + log(Norm3 / delta);

		// Calcula os expoentes globais
		//Lyap[0][idxIteracoes + 1] = (1 / (tempo - t_init)) * LyapSoma[0][idxIteracoes + 1];
		//Lyap[1][idxIteracoes + 1] = (1 / (tempo - t_init)) * LyapSoma[1][idxIteracoes + 1];
		//Lyap[2][idxIteracoes + 1] = (1 / (tempo - t_init)) * LyapSoma[2][idxIteracoes + 1];
		/* ************************************************************************************ */


		/* ************************************************************************************ */
		// INICIALIZACAO DAS CONDICOES INICIAIS PARA A PROXIMA ITERACAO
		// Sistema ORIGINAL
		y10 = Y1;
		y20 = Y2;
		y30 = Y3;

		// CLONES REFERENTE PARA AH VARIAVEL y(1)
		y40 = Y1 + delta*uk[0][0];
		y50 = Y1 + delta*uk[0][1];
		y60 = Y1 + delta*uk[0][2];

		// CLONES REFERENTE PARA AH VARIAVEL y(2)
		y70 = Y2 + delta*uk[1][0];
		y80 = Y2 + delta*uk[1][1];
		y90 = Y2 + delta*uk[1][2];

		// CLONES REFERENTE PARA AH VARIAVEL y(3)
		y100 = Y3 + delta*uk[2][0];
		y110 = Y3 + delta*uk[2][1];
		y120 = Y3 + delta*uk[2][2];
		/* ************************************************************************************ */
		// Atualizoes para a iteracao seguinte
		idxIteracoes++;
		idxIntegra = 0;
		/* ************************************************************************************ */

	} // FIM DO LOOP REFERENTES AHS ITERACOES


	/* ************************************************************************************ */
	// RETORNO DO KERNEL: EXPOENTE GLOBAL

	outLyap1[globalThreadIdx] = LyapSoma[0][0] / (double)(t_final - t_init);
	outLyap2[globalThreadIdx] = LyapSoma[1][0] / (double)(t_final - t_init);
	outLyap3[globalThreadIdx] = LyapSoma[2][0] / (double)(t_final - t_init);

	/* ************************************************************************************ */

} // FIM DO KERNEL
