/*
 * Versao inicial: 27/02/2013
 *
 * Ultima atualizacao: 27/02/2013
 *
 * DESCRICAO:
 * 		- Implementa a integracao do sistema original e clones referente ah analise do modelo
 * 		dinamico do oscilador forcado de Duffing;
 */


#include <math.h>
#include <hip/hip_runtime.h>
	// For the CUDA runtime routines (prefixed with "cuda_")

//#define PI 3.14159265358979323846264338327


/* DESCRICAO:
 * 	- Work out which piece of the global array this thread should operate on;
 * 	- Por ter sido definido como DEVICE, somente pode ser envocado pelo proprio "device", i.e., somente pela GPU;
 * 	- Essa funcao foi definida para facilitar a manipulacao dos indices envolvidos dentro do kernel principal;
 * 	- Utiliza como base o script originalmente desenvolvido para a iteracao do conjunto de Mandelbrot (MATLAB);
 */
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    //	- As entradas serao matrizes;
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}




/* DESCRICAO:
 *  - Main entry point;
 *  - Works out where the current thread should read/write to global memory and calls doIterations to do the actual work.
 */
__global__ void processDuffing1989RK4_ClDyn(
                      double * Y1out, double * Y2out, double * Y3out,
                      double * Y4out, double * Y5out, double * Y6out,
                      double * Y7out, double * Y8out, double * Y9out,
                      double * Y10out, double * Y11out, double * Y12out,
                      const double * Y0_var1, const double * Y0_var2, const double * Y0_var3,
                      const double * Y0_var4, const double * Y0_var5, const double * Y0_var6,
                      const double * Y0_var7, const double * Y0_var8, const double * Y0_var9,
                      const double * Y0_var10, const double * Y0_var11, const double * Y0_var12,
                      const double * H,
                      const double * Gamma,
                      const double * Epsilon,
                      const double * Omega,
                      const unsigned int nMaxItera, 
                      const unsigned int numel ) {
	
	/* **************************************************************************************************** */
	// ALOCA MEMORIA PARA AS VARIAVEIS NECESSARIAS AO CALCULO DO INTEGRADOR 'RK4'

	// Coeficientes parciais para o sistema ORIGINAL
	double k1y1, k2y1, k3y1, k4y1;
	double k1y2, k2y2, k3y2, k4y2;
	double k1y3, k2y3, k3y3, k4y3;

	// Coeficientes parciais para os sistemas CLONADOS
	double k1y4, k2y4, k3y4, k4y4;		// clones referente variavel y(1)
	double k1y5, k2y5, k3y5, k4y5;
	double k1y6, k2y6, k3y6, k4y6;

	double k1y7, k2y7, k3y7, k4y7;		// clones referente variavel y(2)
	double k1y8, k2y8, k3y8, k4y8;
	double k1y9, k2y9, k3y9, k4y9;

	double k1y10, k2y10, k3y10, k4y10;	// clones referente variavel y(3)
	double k1y11, k2y11, k3y11, k4y11;
	double k1y12, k2y12, k3y12, k4y12;


	// Coeficientes finais para o sistema ORIGINAL
	double ky1, ky2, ky3;

	// Coeficientes finais para os sistemas CLONADOS
	double ky4, ky5, ky6;		// clones referente variavel y(1)
	double ky7, ky8, ky9;		// clones referente variavel y(2)
	double ky10, ky11, ky12;	// clones referente variavel y(3)


	// Variaveis temporarias referente ah atualizacao dos estados
	double Y1, Y2, Y3;		// sistema ORIGINAL

	double Y4, Y5, Y6;		// clones referente variavel y(1)
	double Y7, Y8, Y9;		// clones referente variavel y(2)
	double Y10, Y11, Y12;	// clones referente variavel y(3)
	/* **************************************************************************************************** */
	// INICIALIZA VARIAVEIS AUXILIARES DIVERSAS
	unsigned int idxItera = 0;
	/* **************************************************************************************************** */
	// AQUISICAO DOS INDICES REFERENTES AHS THREADS E BLOCOS

	// Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    
    // If we're off the end, return now
    if (globalThreadIdx >= numel) {
        return;
    }
    /* **************************************************************************************************** */
    // INICIALIZACAO DOS PARAMETROS DO SISTEMA DINAMICO
    double h = H[globalThreadIdx];
    double epsilon = Epsilon[globalThreadIdx];
    double gamma   = Gamma[globalThreadIdx];
    double omega 	= Omega[globalThreadIdx];
    /* **************************************************************************************************** */
    // INICIALIZACAO DAS CONDICOES INICIAIS

	// Sistema ORIGINAL
    double y10 = Y0_var1[globalThreadIdx];
    double y20 = Y0_var2[globalThreadIdx];
    double y30 = Y0_var3[globalThreadIdx];

    // CLONES REFERENTE PARA AH VARIAVEL y(1)
    double y40 = Y0_var4[globalThreadIdx];
    double y50 = Y0_var5[globalThreadIdx];
    double y60 = Y0_var6[globalThreadIdx];

    // CLONES REFERENTE PARA AH VARIAVEL y(2)
    double y70 = Y0_var7[globalThreadIdx];
    double y80 = Y0_var8[globalThreadIdx];
    double y90 = Y0_var9[globalThreadIdx];

    // CLONES REFERENTE PARA AH VARIAVEL y(3)
    double y100 = Y0_var10[globalThreadIdx];
    double y110 = Y0_var11[globalThreadIdx];
    double y120 = Y0_var12[globalThreadIdx];
    /* **************************************************************************************************** */
    
    

    
    while ( idxItera < nMaxItera ){
    	/* -------------------------------------------------------------------------------- */
    	// 1o. PASSO
    	// 		- COEFICIENTES PARCIAIS PARA O SISTEMA ORIGINAL
    	k1y1 = h*( y20 );
    	k1y2 = h*( y10 - pow(y10,3) - epsilon*y20 + gamma*cos( omega*y30 ) );
    	k1y3 = h*( 1 );

    	k2y1 = h*( (y20+0.5*k1y2) );
    	k2y2 = h*( (y10+0.5*k1y1) - pow((y10+0.5*k1y1),3) - epsilon*(y20+0.5*k1y2) + gamma*cos( omega*(y30+0.5*k1y3) ) );
    	k2y3 = h*( 1 );

    	k3y1 = h*( (y20+0.5*k2y2) );
    	k3y2 = h*( (y10+0.5*k2y1) - pow((y10+0.5*k2y1),3) - epsilon*(y20+0.5*k2y2) + gamma*cos( omega*(y30+0.5*k2y3) ) );
    	k3y3 = h*( 1 );

    	k4y1 = h*( (y20+k3y2) );
    	k4y2 = h*( (y10+k3y1) - pow((y10+k3y1),3) - epsilon*(y20+k3y2) + gamma*cos( omega*(y30+k3y3) ) );
    	k4y3 = h*( 1 );


    	// 		- COEFICIENTES PARCIAIS PARA OS CLONES
    	k1y4 = h*( y70 );
    	k1y5 = h*( y80 );
    	k1y6 = h*( y90 );
    	k1y7 = h*( y40 - pow(y40,3) - epsilon*y70 + gamma*cos( omega*y100 ) );
    	k1y8 = h*( y50 - pow(y50,3) - epsilon*y80 + gamma*cos( omega*y110 ) );
    	k1y9 = h*( y60 - pow(y60,3) - epsilon*y90 + gamma*cos( omega*y120 ) );
    	k1y10 = h*( 1 );
    	k1y11 = h*( 1 );
    	k1y12 = h*( 1 );

    	k2y4 = h*( (y70+0.5*k1y7) );
    	k2y5 = h*( (y80+0.5*k1y8) );
    	k2y6 = h*( (y90+0.5*k1y9) );
    	k2y7 = h*( (y40+0.5*k1y4) - pow((y40+0.5*k1y4),3) - epsilon*(y70+0.5*k1y7) + gamma*cos( omega*(y100+0.5*k1y10) ) );
    	k2y8 = h*( (y50+0.5*k1y5) - pow((y50+0.5*k1y5),3) - epsilon*(y80+0.5*k1y8) + gamma*cos( omega*(y110+0.5*k1y11) ) );
    	k2y9 = h*( (y60+0.5*k1y6) - pow((y60+0.5*k1y6),3) - epsilon*(y90+0.5*k1y9) + gamma*cos( omega*(y120+0.5*k1y12) ) );
    	k2y10 = h*( 1 );
    	k2y11 = h*( 1 );
    	k2y12 = h*( 1 );

    	k3y4 = h*( (y70+0.5*k2y7) );
    	k3y5 = h*( (y80+0.5*k2y8) );
    	k3y6 = h*( (y90+0.5*k2y9) );
    	k3y7 = h*( (y40+0.5*k2y4) - pow((y40+0.5*k2y4),3) - epsilon*(y70+0.5*k2y7) + gamma*cos( omega*(y100+0.5*k2y10) ) );
    	k3y8 = h*( (y50+0.5*k2y5) - pow((y50+0.5*k2y5),3) - epsilon*(y80+0.5*k2y8) + gamma*cos( omega*(y110+0.5*k2y11) ) );
    	k3y9 = h*( (y60+0.5*k2y6) - pow((y60+0.5*k2y6),3) - epsilon*(y90+0.5*k2y9) + gamma*cos( omega*(y120+0.5*k2y12) ) );
    	k3y10 = h*( 1 );
    	k3y11 = h*( 1 );
    	k3y12 = h*( 1 );

    	k4y4 = h*( (y70+k3y7) );
    	k4y5 = h*( (y80+k3y8) );
    	k4y6 = h*( (y90+k3y9) );
    	k4y7 = h*( (y40+k3y4) - pow((y40+k3y4),3) - epsilon*(y70+k3y7) + gamma*cos( omega*(y100+k3y10) ) );
    	k4y8 = h*( (y50+k3y5) - pow((y50+k3y5),3) - epsilon*(y80+k3y8) + gamma*cos( omega*(y110+k3y11) ) );
    	k4y9 = h*( (y60+k3y6) - pow((y60+k3y6),3) - epsilon*(y90+k3y9) + gamma*cos( omega*(y120+k3y12) ) );
    	k4y10 = h*( 1 );
    	k4y11 = h*( 1 );
    	k4y12 = h*( 1 );
    	/* -------------------------------------------------------------------------------- */
    	// 2o. PASSO
    	// 		- COEFICIENTES PARA O SISTEMA ORIGINAL
    	ky1 = (1/6.0)*( k1y1 + 2*k2y1 + 2*k3y1 + k4y1 );
    	ky2 = (1/6.0)*( k1y2 + 2*k2y2 + 2*k3y2 + k4y2 );
    	ky3 = (1/6.0)*( k1y3 + 2*k2y3 + 2*k3y3 + k4y3 );

    	// 		- COEFICIENTES PARA OS CLONES
    	ky4 = (1/6.0)*( k1y4 + 2*k2y4 + 2*k3y4 + k4y4 );
    	ky5 = (1/6.0)*( k1y5 + 2*k2y5 + 2*k3y5 + k4y5 );
    	ky6 = (1/6.0)*( k1y6 + 2*k2y6 + 2*k3y6 + k4y6 );

    	ky7 = (1/6.0)*( k1y7 + 2*k2y7 + 2*k3y7 + k4y7 );
    	ky8 = (1/6.0)*( k1y8 + 2*k2y8 + 2*k3y8 + k4y8 );
    	ky9 = (1/6.0)*( k1y9 + 2*k2y9 + 2*k3y9 + k4y9 );

    	ky10 = (1/6.0)*( k1y10 + 2*k2y10 + 2*k3y10 + k4y10 );
    	ky11 = (1/6.0)*( k1y11 + 2*k2y11 + 2*k3y11 + k4y11 );
    	ky12 = (1/6.0)*( k1y12 + 2*k2y12 + 2*k3y12 + k4y12 );
    	/* -------------------------------------------------------------------------------- */
    	// 3o. PASSO:
    	//   - Atualizacao do sistema (aplicacao do PASSO de Euler);

    	// SISTEMA ORIGINAL
    	Y1 = y10 + ky1;
    	Y2 = y20 + ky2;
    	Y3 = y30 + ky3;

    	// CLONES
    	Y4 = y40 + ky4;
    	Y5 = y50 + ky5;
    	Y6 = y60 + ky6;

    	Y7 = y70 + ky7;
    	Y8 = y80 + ky8;
    	Y9 = y90 + ky9;

    	Y10 = y100 + ky10;
    	Y11 = y110 + ky11;
    	Y12 = y120 + ky12;
    	/* -------------------------------------------------------------------------------- */
    	// ATUALIZACAO DAS CONDICOES PARA A PROXIMA ITERACAO
    	y10 = Y1;
    	y20 = Y2;
    	y30 = Y3;

    	y40 = Y4;
    	y50 = Y5;
    	y60 = Y6;

    	y70 = Y7;
    	y80 = Y8;
    	y90 = Y9;

    	y100 = Y10;
    	y110 = Y11;
    	y120 = Y12;
    	/* -------------------------------------------------------------------------------- */
    	// Atualiza o contador
    	idxItera++;
    	/* -------------------------------------------------------------------------------- */
    }
    
    // Retorna o estado final do sistema apos o numero de iteracoes maximo considerado
    Y1out[globalThreadIdx] = Y1;
    Y2out[globalThreadIdx] = Y2;
    Y3out[globalThreadIdx] = Y3;

    Y4out[globalThreadIdx] = Y4;
    Y5out[globalThreadIdx] = Y5;
    Y6out[globalThreadIdx] = Y6;

    Y7out[globalThreadIdx] = Y7;
    Y8out[globalThreadIdx] = Y8;
    Y9out[globalThreadIdx] = Y9;

    Y10out[globalThreadIdx] = Y10;
    Y11out[globalThreadIdx] = Y11;
    Y12out[globalThreadIdx] = Y12;
}
