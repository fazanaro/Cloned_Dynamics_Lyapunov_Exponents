/* ***************************************************************************************************************
 * 
 * VERSAO INICIAL: 12/02/2014
 * 
 * ULTIMA ATUALIZACAO: 19/02/2014
 * 
 * ***************************************************************************************************************
 * 
 * DESCRICAO:
 * 	- Constroi a secao de Poincare, tomando como base a amostragem da variavel de estado 'Z', para o modelo do
 * 	oscilador forcado de Duffing;
 * 	
 * 	- FAZ USO DA CARACTERISTICA DA ALOCACAO DE MEMORIA EM C (ROW-MAJOR LAYOUT): AS LINHAS DE UMA MATRIZ NXM PODEM
 * 	SER ALINHADAS NO FORMATO DE UM VETOR DE COMPRIMENTO TOTAL N*M (ver pg 71, sec 4.2, [Kirk, Hwu (2013)]);
 *
 * 	- Assim sendo, esse kernel retorna um vetor de comprimento igual ah nMaxPoincare*length(vEpsilon). NOTE QUE
 * 	A INDICACAO DE QUANDO SE INICIA UM NOVO TRECHO CORRESPONDENTE AH MODIFICACAO DO PARAMETRO DE CONTROLE EH
 * 	DEFINIDA PELA POSICAO DA THREAD;
 * 	
 * ***************************************************************************************************************
 * 
 * REFERENCIAS:
 * 
 * 	http://www.cprogramming.com/tutorial/cfileio.html
 * 	
 * 	http://stackoverflow.com/questions/11573974/write-to-txt-file
 * 	
 * *************************************************************************************************************** 	 
 */

#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
	// For the CUDA runtime routines (prefixed with "cuda_")

//#define PI 3.14159265358979323846264338327







/* *************************************************************************************************************** */
/* DESCRICAO:
 * 	- Work out which piece of the global array this thread should operate on;
 * 	- Por ter sido definido como DEVICE, somente pode ser envocado pelo proprio "device", i.e., somente pela GPU;
 * 	- Essa funcao foi definida para facilitar a manipulacao dos indices envolvidos dentro do kernel principal;
 * 	- Utiliza como base o script originalmente desenvolvido para a iteracao do conjunto de Mandelbrot (MATLAB);
 */

__device__ size_t calculateGlobalIndex() {
	// Which block are we?
	size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
	// Which thread are we within the block?
	size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
	// How big is each block?
	//	- As entradas serao matrizes;
	size_t const threadsPerBlock = blockDim.x*blockDim.y;
	// Which thread are we overall?
	return localThreadIdx + globalBlockIndex*threadsPerBlock;
}

/* *************************************************************************************************************** */

/* DESCRICAO:
 *  - Main entry point;
 *  - Works out where the current thread should read/write to global memory and calls doIterations to do the actual work.
 */
__global__ void Duffing_PoincareZ(
		double * X,
		double * Y,
		const double * X0,
		const double * Y0,
		const double * Z0,
		const double * H,
		const double * Gamma,
		const double * Epsilon,
		const double * Omega,
		const double Threshold,
		const unsigned int numel ) {

	/* **************************************************************************************** */
	// AQUISICAO DOS INDICES REFERENTES AHS THREADS E BLOCOS

	// Work out which thread we are
	size_t const globalThreadIdx = calculateGlobalIndex();

	// If we're off the end, return now
	if (globalThreadIdx >= numel) {
		return;
	}

	/* **************************************************************************************** */
	// DEFINE O NUMERO MAXIMO DE PONTOS QUE ESTARAO PRESENTES NA SECAO

	const int numMaxElementsPoincare = 5;

	/* **************************************************************************************** */
	// INICIALIZACAO DOS PARAMETROS DO SISTEMA DINAMICO

	double epsilon = Epsilon[globalThreadIdx];
	double gamma 	= Gamma[globalThreadIdx];
	double omega 	= Omega[globalThreadIdx];

	/* **************************************************************************************** */
	// DEFINE O PASSO DE INCREMENTO DO INTEGRADOR

	const double h = H[globalThreadIdx];

	/* **************************************************************************************** */
	// CARREGA AS CONDICOES INICIAIS PASSADAS AO KERNEL PELO SCRIPT EM MATLAB

	double x0 = X0[globalThreadIdx];
	double y0 = Y0[globalThreadIdx];
	double z0 = Z0[globalThreadIdx];

	/* **************************************************************************************** */
	// DEFINICAO DAS CONDICOES INICIAIS E OPCOES DE INTEGRACAO

	// Instante inicial de integracao
	const double t_init = 0;

	// Define o instante de tempo final de simulacao
	//const double t_final = 30.0;

	// Define o tempo de transitorio
	const double t_trans = 10.0;

	// Define o indice do transitorio, ou seja, os pontos que deverao ser desconsiderados
	// 		- Esse valor eh estimado a partir da construcao da secao de Poincare
	const int index_trans = round( (t_trans-t_init)/h );

	/* **************************************************************************************** */
	// ALOCA MEMORIA

	// Inicializa a variavel que representa o numero de pontos adquiridos
	int numPtosPoincareAdquiridos = 0;


	// Inicializacao do vetor que contem os indices do pontos que estao sobre
	// (ou muito proximos) da secao de Poincare
	int index[numMaxElementsPoincare];


	// Aloca memoria para os vetores que definem a secao de Poincare
	double vPoincareX[numMaxElementsPoincare];
	double vPoincareY[numMaxElementsPoincare];


	// Aloca memoria para os vetores auxiliares
	double varX[2], varY[2], varZ[2];

	double 	vAmostragem[2];

	/* **************************************************************************************** */
	// INICIALIZA OS PARAMETROS PARA A INTERPOLACAO DOS DADOS

	double CoefAngular;
	double CoefLinear;

	/* **************************************************************************************** */
	// INICIALIZA A VARIAVEL DE CONTAGEM PARA A INTEGRACAO DO MODELO DINAMICO

	unsigned int numContagemIteracoes = 0;

	/* **************************************************************************************** */
	// INICIALIZACAO DAS CONDICOES INICIAIS E DOS COEFICIENTES DO ALGORITMO RK4

	double k1x, k2x, k3x, k4x;
	double k1y, k2y, k3y, k4y;
	double k1z, k2z, k3z, k4z;

	double x, y, z;

	/* **************************************************************************************** */
	// INTEGRA O MODELO DINAMICO

	while (numPtosPoincareAdquiridos <= numMaxElementsPoincare){

		// ==================================================================================== //
		//							INTEGRACAO DO MODELO DINAMICO								//
		// ==================================================================================== //
		// 1o. PASSO: 	CALCULA OS COEFICIENTES PARCIAIS

		k1x = h*( y0 );
		k1y = h*( x0 - pow(x0,3) - epsilon*y0 + gamma*cos( omega*z0 ) );
		k1z = h*( 1 );

		k2x = h*( (y0+0.5*k1y) );
		k2y = h*( (x0+0.5*k1x) - pow((x0+0.5*k1x),3) - epsilon*(y0+0.5*k1y) + gamma*cos( omega*(z0+0.5*k1z) ) );
		k2z = h*( 1 );

		k3x = h*( (y0+0.5*k2y) );
		k3y = h*( (x0+0.5*k2x) - pow((x0+0.5*k2x),3) - epsilon*(y0+0.5*k2y) + gamma*cos( omega*(z0+0.5*k2z) ) );
		k3z = h*( 1 );

		k4x = h*( (y0+k3y) );
		k4y = h*( (x0+k3x) - pow((x0+k3x),3) - epsilon*(y0+k3y) + gamma*cos( omega*(z0+k3z) ) );
		k4z = h*( 1 );

		/* ----------------------------------------------------------------------------------- */
		// 2o. PASSO: 	ATUALIZA AS VARIAVEIS DE ESTADO

		x = x0 + (1/6.0)*( k1x + 2*k2x + 2*k3x + k4x );
		y = y0 + (1/6.0)*( k1y + 2*k2y + 2*k3y + k4y );
		z = z0 + (1/6.0)*( k1z + 2*k2z + 2*k3z + k4z );

		/* ----------------------------------------------------------------------------------- */
		// 3o. PASSO: 	ATUALIZA AS VARIAVEIS DE ESTADO PARA A PROXIMA ITERACAO

		//x0 = x;
		//y0 = y;
		//z0 = z;

		// ==================================================================================== //
		// 							ATUALIZA O NUMERO DE ITERACOES								//
		// ==================================================================================== //

		numContagemIteracoes = numContagemIteracoes + 1;

		// ==================================================================================== //
		// 					FAZ A ANALISE DOS PONTOS: INTERPOLACAO DOS DADOS					//
		// ==================================================================================== //

		if (numContagemIteracoes >= 2){
			//if (numContagemIteracoes >= index_trans){

			// -------------------------------------------------------------------------------- //

			varX[0] = x0;
			varY[0] = y0;
			varZ[0] = z0;

			varX[1] = x;
			varY[1] = y;
			varZ[1] = z;

			vAmostragem[0] = sin( omega*varZ[0] );
			vAmostragem[1] = sin( omega*varZ[1] );

			// -------------------------------------------------------------------------------- //

			if (vAmostragem[0] == Threshold){
				// ---------------------------------------------------------------------------- //
				// O PONTO ESTAH EXATAMENTE SOBRE A SECAO DE POINCARE
				// ---------------------------------------------------------------------------- //
				// SALVA OS PONTOS

				vPoincareX[numPtosPoincareAdquiridos] = x0;
				vPoincareY[numPtosPoincareAdquiridos] = y0;

				// ---------------------------------------------------------------------------- //
				// ARMAZENA OS INDICES

				index[numPtosPoincareAdquiridos] = numContagemIteracoes;

				// ---------------------------------------------------------------------------- //
				// ATUALIZA O NUMERO TOTAL DE PONTOS ADQUIRIDOS

				numPtosPoincareAdquiridos = numPtosPoincareAdquiridos + 1;

				// ---------------------------------------------------------------------------- //

			}else if ( (vAmostragem[0] <= Threshold) && (vAmostragem[1] >= Threshold) ){
				// ---------------------------------------------------------------------------- //
				// CONDICAO PARA AMOSTRAGEM NO SENTIDO POSITIVO
				// ---------------------------------------------------------------------------- //
				// Se o ponto nao estah sobre a secao de Poincare, serah verificado se ocorreu
				// cruzamento. Se sim, serah salvo o indice referente ao ponto imediatamente
				// anterior ah secao de Poincare
				// ---------------------------------------------------------------------------- //
				// AMOSTRAGEM DE 'X'

				// Calcula-se os coeficientes da reta que ajusta os pontos
				CoefAngular = ( vAmostragem[1]-vAmostragem[0] ) / ( varX[1]-varX[0] );
				CoefLinear  = ((vAmostragem[1]+vAmostragem[0])  - CoefAngular*( varX[1]+varX[0] ) )/2.0;

				// Calcula os pontos que seriam obtidos caso a secao de Poincare fosse atingida
				// de maneira ideal
				vPoincareX[numPtosPoincareAdquiridos] = (Threshold - CoefLinear)/CoefAngular;

				// ---------------------------------------------------------------------------- //
				// AMOSTRAGEM DE 'Y'

				// Calcula-se os coeficientes da reta que ajusta os pontos
				CoefAngular = ( vAmostragem[1]-vAmostragem[0] ) / ( varY[1]-varY[0] );
				CoefLinear  = ((vAmostragem[1]+vAmostragem[0])  - CoefAngular*( varY[1]+varY[0] ) )/2.0;

				// Calcula os pontos que seriam obtidos caso a secao de Poincare fosse atingida
				// de maneira ideal
				vPoincareY[numPtosPoincareAdquiridos] = (Threshold - CoefLinear)/CoefAngular;

				// ---------------------------------------------------------------------------- //
				// ARMAZENA OS INDICES

				index[numPtosPoincareAdquiridos] = numContagemIteracoes;

				// ---------------------------------------------------------------------------- //
				// ATUALIZA O NUMERO TOTAL DE PONTOS ADQUIRIDOS

				numPtosPoincareAdquiridos = numPtosPoincareAdquiridos + 1;

				// ---------------------------------------------------------------------------- //

			}

		}

		// ==================================================================================== //
		// 					ATUALIZA AS CONDICOES PARA A PROXIMA ITERACAO						//
		// ==================================================================================== //

		x0 = x;
		y0 = y;
		z0 = z;

		// ==================================================================================== //

	}

	/* **************************************************************************************** */
	//								RETORNA A SAIDA DO KERNEL
	/* **************************************************************************************** */

	for (int ii = 0; ii < numMaxElementsPoincare; ii++){

		X[globalThreadIdx*numMaxElementsPoincare + ii] = vPoincareX[ii];
		Y[globalThreadIdx*numMaxElementsPoincare + ii] = vPoincareY[ii];

	}

	/* **************************************************************************************** */

}

